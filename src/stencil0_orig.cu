
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define CUDA_CALL(c) \
    do {                                            \
        hipError_t res = c;                        \
        if (res != hipSuccess) {                   \
            fprintf(stderr, "error at line %d: %s \n", __LINE__, hipGetErrorString(res));    \
            exit(EXIT_FAILURE);                     \
        }                                           \
    } while (0)

// Problem dimension
#define STENCIL_SIZE (16 * 1024 * 1024)
#define RADIUS (3)
#define NUM_CHANNELS (4)

#define BLOCK_SIZE (32)

// Given the pixel index and channel, return the position of the
// element in the 1D array.
static __device__ __host__ int getIndex(int index, int channel)
{
    return index * NUM_CHANNELS + channel;
}

static __constant__ int weight[RADIUS * 2 + 1] = {
    1, 2, 3, 4, 3, 2, 1
};
static __constant__ int denominator = 16;

__global__ void stencilKernel(unsigned char *in,
                              int numPixels,
                              unsigned char *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int radius, channel;
    int outLocal[NUM_CHANNELS] = { 0 };

    if (i < RADIUS) {
        return;
    }

    if (i >= numPixels - RADIUS) {
        return;
    }

    for (radius = -RADIUS; radius <= RADIUS; ++radius) {
        for (channel = 0; channel < NUM_CHANNELS; ++channel) {
            outLocal[channel] += (int)in[getIndex(i + radius, channel)] * weight[RADIUS + radius];
        }
    }

    for (channel = 0; channel < NUM_CHANNELS; ++channel) {
        out[getIndex(i, channel)] = (unsigned char)(outLocal[channel] / denominator);
    }
}

static void stencilGpu(unsigned char *in,
                       int numPixels,
                       unsigned char *out)
{
    unsigned char *inGPU;
    unsigned char *outGPU;
    size_t arraySize;

    arraySize = numPixels * NUM_CHANNELS * sizeof(unsigned char);

    CUDA_CALL(hipMalloc(&inGPU, arraySize));
    CUDA_CALL(hipMalloc(&outGPU, arraySize));

    CUDA_CALL(hipMemcpy(inGPU, in, arraySize, hipMemcpyHostToDevice));

    stencilKernel<<<ceil((float)numPixels / BLOCK_SIZE), BLOCK_SIZE>>>(inGPU, numPixels, outGPU);
    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipDeviceSynchronize());

    CUDA_CALL(hipMemcpy(out, outGPU, arraySize, hipMemcpyDeviceToHost));

    CUDA_CALL(hipFree(inGPU));
    CUDA_CALL(hipFree(outGPU));
}

int main()
{
    unsigned char *in;
    unsigned char *outGPU;
    size_t arraySize = STENCIL_SIZE * NUM_CHANNELS * sizeof(unsigned char);

    in = (unsigned char *)malloc(arraySize);
    outGPU = (unsigned char *)malloc(arraySize);
    if (in == NULL || outGPU == NULL) {
        fprintf(stderr, "Allocation failed\n");
        exit(EXIT_FAILURE);
    }

    stencilGpu(in, STENCIL_SIZE, outGPU);

    free(in);
    free(outGPU);

    return 0;
}
